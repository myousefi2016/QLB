/**
 *	Quantum Lattice Boltzmann 
 *	(c) 2015 Fabian Thüring, ETH Zurich
 *
 *	[DESCRIPTION]
 *	This file contains all the CUDA kernels and device functions
 */

#include "QLB.hpp"
#include "cuassert.hpp"

void QLB::allocate_device_arrays()
{
	//cuassert( hipMalloc((void**)&d_spinor_, sizeof(d_spinor_[0])) );
}	

void QLB::free_device_arrays()
{
	//cuassert( hipFree(d_spinor_) );
	//hipDeviceReset();
}

void QLB::init_device()
{
	// Print CUDA informations if requested
	if(verbose_)
	{
		std::cout << " === CUDA Info === " << std::endl;
		hipDeviceProp_t deviceProp; hipGetDeviceProperties(&deviceProp, 0);
		int dvVers = 0; hipDriverGetVersion(&dvVers);
		int rtVers = 0; hipRuntimeGetVersion(&rtVers);
		unsigned mem = (unsigned)deviceProp.totalGlobalMem;
		std::printf("CUDA Driver Version:  %d.%d\n", dvVers/1000, dvVers%100);
		std::printf("CUDA Runtime Version: %d.%d\n", rtVers/1000, rtVers%100);
		std::printf("Total GPU memory:     %u bytes\n", mem);
	}

	//cuassert( hipMemcpy(d_spinor_, spinor1_.data() , sizeof(spinor1_[0]), hipMemcpyHostToDevice) );
	//cuassert( hipMemcpy(d_spinor_, X.data() , sizeof(X[0]), hipMemcpyHostToDevice) );
}

void QLB::get_device_arrays()
{
	//cuassert( hipMemcpy(spinor1_.data() , d_spinor_, sizeof(spinor1_[0]), hipMemcpyDeviceToHost) );
}
